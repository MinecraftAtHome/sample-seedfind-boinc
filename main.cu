#include "hip/hip_runtime.h"
#include <string.h>
#include <stdio.h>
#include <inttypes.h>
#include <math.h>



#define __STDC_FORMAT_MACROS 1

#include <stdlib.h>
#include <stddef.h>
#include <inttypes.h>

#ifdef BOINC
  #include "boinc_api.h"
#if defined _WIN32 || defined _WIN64
  #include "boinc_win.h"
#endif
#endif

#define GPU_ASSERT(code) gpuAssert((code), __FILE__, __LINE__)
inline void gpuAssert(hipError_t code, const char *file, int line) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s (code %d) %s %d\n", hipGetErrorString(code), code, file, line);
    exit(code);
  }
}

///=============================================================================
///                      Compiler and Platform Features
///=============================================================================

typedef int8_t      i8;
typedef uint8_t     u8;
typedef int16_t     i16;
typedef uint16_t    u16;
typedef int32_t     i32;
typedef uint32_t    u32;
typedef int64_t     i64;
typedef uint64_t    u64;
typedef float       f32;
typedef double      f64;


#define STRUCT(S) typedef struct S S; struct S

#if __GNUC__

#define IABS(X)                 __builtin_abs(X)
#define PREFETCH(PTR,RW,LOC)    __builtin_prefetch(PTR,RW,LOC)
#define likely(COND)            (__builtin_expect(!!(COND),1))
#define unlikely(COND)          (__builtin_expect((COND),0))
#define ATTR(...)               __attribute__((__VA_ARGS__))
#define BSWAP32(X)              __builtin_bswap32(X)
#define UNREACHABLE()           __builtin_unreachable()

#else

#define IABS(X)                 ((int)abs(X))
#define PREFETCH(PTR,RW,LOC)
#define likely(COND)            (COND)
#define unlikely(COND)          (COND)
#define ATTR(...)
__device__ __host__ static inline uint32_t BSWAP32(uint32_t x) {
    x = ((x & 0x000000ff) << 24) | ((x & 0x0000ff00) <<  8) |
        ((x & 0x00ff0000) >>  8) | ((x & 0xff000000) >> 24);
    return x;
}
#if _MSC_VER
#define UNREACHABLE()           
#else
#define UNREACHABLE()           exit(1) // [[noreturn]]
#endif

#endif

/// imitate amd64/x64 rotate instructions

__device__ __host__ static inline ATTR(const, always_inline, artificial)
uint64_t rotl64(uint64_t x, uint8_t b)
{
    return (x << b) | (x >> (64-b));
}

__device__ __host__ static inline ATTR(const, always_inline, artificial)
uint32_t rotr32(uint32_t a, uint8_t b)
{
    return (a >> b) | (a << (32-b));
}

/// integer floor divide
__device__ __host__ static inline ATTR(const, always_inline)
int32_t floordiv(int32_t a, int32_t b)
{
    int32_t q = a / b;
    int32_t r = a % b;
    return q - ((a ^ b) < 0 && !!r);
}

///=============================================================================
///                    C implementation of Java Random
///=============================================================================

__device__ __host__ static inline void setSeed(uint64_t *seed, uint64_t value)
{
    *seed = (value ^ 0x5deece66d) & ((1ULL << 48) - 1);
}

__device__ __host__ static inline int next(uint64_t *seed, const int bits)
{
    *seed = (*seed * 0x5deece66d + 0xb) & ((1ULL << 48) - 1);
    return (int) ((int64_t)*seed >> (48 - bits));
}

__device__ __host__ static inline int nextInt(uint64_t *seed, const int n)
{
    int bits, val;
    const int m = n - 1;

    if ((m & n) == 0) {
        uint64_t x = n * (uint64_t)next(seed, 31);
        return (int) ((int64_t) x >> 31);
    }

    do {
        bits = next(seed, 31);
        val = bits % n;
    }
    while (bits - val + m < 0);
    return val;
}

__device__ __host__ static inline uint64_t nextLong(uint64_t *seed)
{
    return ((uint64_t) next(seed, 32) << 32) + next(seed, 32);
}

__device__ __host__ static inline float nextFloat(uint64_t *seed)
{
    return next(seed, 24) / (float) (1 << 24);
}

__device__ __host__ static inline double nextDouble(uint64_t *seed)
{
    uint64_t x = (uint64_t)next(seed, 26);
    x <<= 27;
    x += next(seed, 27);
    return (int64_t) x / (double) (1ULL << 53);
}

/* A macro to generate the ideal assembly for X = nextInt(*S, 24)
 * This is a macro and not an inline function, as many compilers can make use
 * of the additional optimisation passes for the surrounding code.
 */
#define JAVA_NEXT_INT24(S,X)                \
    do {                                    \
        uint64_t a = (1ULL << 48) - 1;      \
        uint64_t c = 0x5deece66dULL * (S);  \
        c += 11; a &= c;                    \
        (S) = a;                            \
        a = (uint64_t) ((int64_t)a >> 17);  \
        c = 0xaaaaaaab * a;                 \
        c = (uint64_t) ((int64_t)c >> 36);  \
        (X) = (int)a - (int)(c << 3) * 3;   \
    } while (0)


/* Jumps forwards in the random number sequence by simulating 'n' calls to next.
 */
__device__ __host__ static inline void skipNextN(uint64_t *seed, uint64_t n)
{
    uint64_t m = 1;
    uint64_t a = 0;
    uint64_t im = 0x5deece66dULL;
    uint64_t ia = 0xb;
    uint64_t k;

    for (k = n; k; k >>= 1)
    {
        if (k & 1)
        {
            m *= im;
            a = im * a + ia;
        }
        ia = (im + 1) * ia;
        im *= im;
    }

    *seed = *seed * m + a;
    *seed &= 0xffffffffffffULL;
}


///=============================================================================
///                               Xoroshiro 128
///=============================================================================

STRUCT(Xoroshiro)
{
    uint64_t lo, hi;
};

__device__ __host__ static inline void xSetSeed(Xoroshiro *xr, uint64_t value)
{
    const uint64_t XL = 0x9e3779b97f4a7c15ULL;
    const uint64_t XH = 0x6a09e667f3bcc909ULL;
    const uint64_t A = 0xbf58476d1ce4e5b9ULL;
    const uint64_t B = 0x94d049bb133111ebULL;
    uint64_t l = value ^ XH;
    uint64_t h = l + XL;
    l = (l ^ (l >> 30)) * A;
    h = (h ^ (h >> 30)) * A;
    l = (l ^ (l >> 27)) * B;
    h = (h ^ (h >> 27)) * B;
    l = l ^ (l >> 31);
    h = h ^ (h >> 31);
    xr->lo = l;
    xr->hi = h;
}

__device__ __host__ static inline uint64_t xNextLong(Xoroshiro *xr)
{
    uint64_t l = xr->lo;
    uint64_t h = xr->hi;
    uint64_t n = rotl64(l + h, 17) + l;
    h ^= l;
    xr->lo = rotl64(l, 49) ^ h ^ (h << 21);
    xr->hi = rotl64(h, 28);
    return n;
}

__device__ __host__ static inline int xNextInt(Xoroshiro *xr, uint32_t n)
{
    uint64_t r = (xNextLong(xr) & 0xFFFFFFFF) * n;
    if ((uint32_t)r < n)
    {
        while ((uint32_t)r < (~n + 1) % n)
        {
            r = (xNextLong(xr) & 0xFFFFFFFF) * n;
        }
    }
    return r >> 32;
}

__device__ __host__ static inline double xNextDouble(Xoroshiro *xr)
{
    return (xNextLong(xr) >> (64-53)) * 1.1102230246251565E-16;
}

__device__ __host__ static inline float xNextFloat(Xoroshiro *xr)
{
    return (xNextLong(xr) >> (64-24)) * 5.9604645E-8F;
}

__device__ __host__ static inline void xSkipN(Xoroshiro *xr, int count)
{
    while (count --> 0)
        xNextLong(xr);
}

__device__ __host__ static inline uint64_t xNextLongJ(Xoroshiro *xr)
{
    int32_t a = xNextLong(xr) >> 32;
    int32_t b = xNextLong(xr) >> 32;
    return ((uint64_t)a << 32) + b;
}

__device__ __host__ static inline int xNextIntJ(Xoroshiro *xr, uint32_t n)
{
    int bits, val;
    const int m = n - 1;

    if ((m & n) == 0) {
        uint64_t x = n * (xNextLong(xr) >> 33);
        return (int) ((int64_t) x >> 31);
    }

    do {
        bits = (xNextLong(xr) >> 33);
        val = bits % n;
    }
    while (bits - val + m < 0);
    return val;
}


__global__ void kernel(uint64_t s, uint64_t *out) {
    uint64_t input_seed = blockDim.x * blockIdx.x + threadIdx.x + s;
    //Do something to seed, usually best to keep 'input_seed' alone in case you need it later.
    uint64_t seed = input_seed;
    /*Insert filter code here - return early if the seed does not match all criteria*/
    out[blockDim.x * blockIdx.x + threadIdx.x] = seed;
}

#include <time.h>
#include <chrono>
using namespace std::chrono;

#ifdef __GNUC__

#include <unistd.h>
#include <sys/time.h>

#endif

/*
    You can add anything you want to checkpoint_vars.
    Be sure to update the checkpointing sections below to reflect the new item in the struct (to save the data into the struct and then to disk)
*/
struct checkpoint_vars {
    unsigned long long offset;
    uint64_t elapsed_chkpoint;
};

uint64_t elapsed_chkpoint = 0;

int main(int argc, char **argv) {

    /*
        The way this has been written, each loop, it calls 32768 * 32 (1048576) kernel threads that each individually run a single seed.
        We refer to these loops as "blocks" of seeds in this code.
        --start defines the starting block (--start 0 begins at seed 0, --start 1 begins at seed 1048576, --start 2 begins at 2097152)
        --end defines the ending block (--end 0 finishes at seed 0, --end 1 finishes at seed 1048576, --end 3 begins at seed 2097152)
        --device defines which GPU ID runs the cuda kernels. You can check this using nvidia-smi if you're running standalone. Otherwise, if you're running on BOINC, this parameter is unneeded on modern clients. Keep it implemented for old clients.
    */
    uint64_t block_min = 0;
    uint64_t block_max = 0;
    uint64_t checked = 0;
    int device = 0;
    for (int i = 1; i < argc; i += 2) {
		const char *param = argv[i];
		if (strcmp(param, "-d") == 0 || strcmp(param, "--device") == 0) {
			device = atoi(argv[i + 1]);
		} else if (strcmp(param, "-s") == 0 || strcmp(param, "--start") == 0) {
			sscanf(argv[i + 1], "%llu", &block_min);
		} else if (strcmp(param, "-e") == 0 || strcmp(param, "--end") == 0) {
			sscanf(argv[i + 1], "%llu", &block_max);
		} 
        else {
			fprintf(stderr,"Unknown parameter: %s\n", param);
        }
    }
    uint64_t offsetStart = 0;
    uint64_t *out;
    //GPU Params
	int blocks = 32768;
	int threads = 32;
    //BOINC
  	#ifdef BOINC

        BOINC_OPTIONS options;
        boinc_options_defaults(options);
	    options.normal_thread_priority = true;
        boinc_init_options(&options);
        APP_INIT_DATA aid;
	    boinc_get_init_data(aid);
        if (aid.gpu_device_num >= 0) {
            //If BOINC client provided us a device ID
		    device = aid.gpu_device_num;
		    fprintf(stderr,"boinc gpu %i gpuindex: %i \n", aid.gpu_device_num, device);
		} else {
            //If BOINC client did not provide us a device ID
            device = -5;
            for (int i = 1; i < argc; i += 2) {
                //Check for a --device flag, just in case we missed it earlier, use it if it's available. For older clients primarily.
              	if(strcmp(argv[i], "--device") == 0){
                    sscanf(argv[i + 1], "%i", &device);
                }
  
            }
            if(device == -5){
                //Something has gone wrong. It pulled from BOINC, got -1. No --device parameter present.
                fprintf(stderr, "Error: No --device parameter provided! Defaulting to device 0...\n");
                device = 0;
            }
		    fprintf(stderr,"stndalone gpuindex %i (aid value: %i)\n", device, aid.gpu_device_num);
	    }   

        FILE *checkpoint_data = boinc_fopen("checkpoint.txt", "rb");
        if(!checkpoint_data){
            //No checkpoint file was found. Proceed from the beginning.
            fprintf(stderr, "No checkpoint to load\n");

        }
        else{
            //Load from checkpoint. You can put any data in data_store that you need to keep between runs of this program.
            boinc_begin_critical_section();
            struct checkpoint_vars data_store;
            fread(&data_store, sizeof(data_store), 1, checkpoint_data);
            offsetStart = data_store.offset;
            elapsed_chkpoint = data_store.elapsed_chkpoint;
            fprintf(stderr, "Checkpoint loaded, task time %d s, seed pos: %llu\n", elapsed_chkpoint, offsetStart);
            fclose(checkpoint_data);
            boinc_end_critical_section();
        }
    #endif
    hipSetDevice(device);
    hipMallocManaged(&out, (blocks * threads) * sizeof(*out));
    for(int i = 0; i < (blocks * threads); i++){
        out[i] = 0;
    }
    auto start = high_resolution_clock::now();
	printf("starting...\n");
    uint64_t checkpointTemp = 0;
    FILE* seedsout = fopen("seeds.txt", "w+");
    for (uint64_t s = (uint64_t)block_min + offsetStart; s < (uint64_t)block_max; s++) {
        //Call GPU kernel
        kernel<<<blocks, threads>>>(blocks * threads * s, out);
        GPU_ASSERT(hipPeekAtLastError());
        GPU_ASSERT(hipDeviceSynchronize());  
        //Check error from GPU driver, if any
        checkpointTemp += 1;
        #ifdef BOINC
        if(checkpointTemp >= 15 || boinc_time_to_checkpoint()){
            //Checkpointing for BOINC
            auto checkpoint_end = high_resolution_clock::now();
            auto duration = duration_cast<milliseconds>(checkpoint_end - start);
            boinc_begin_critical_section(); // Boinc should not interrupt this
            
            // Checkpointing section below
            boinc_delete_file("checkpoint.txt"); // Don't touch, same func as normal fdel
            FILE *checkpoint_data = boinc_fopen("checkpoint.txt", "wb");
            struct checkpoint_vars data_store;
            data_store.offset = s - block_min;
            data_store.elapsed_chkpoint = elapsed_chkpoint + duration.count();
            fwrite(&data_store, sizeof(data_store), 1, checkpoint_data);
            fclose(checkpoint_data);
            checkpointTemp = 0;
            boinc_end_critical_section();
            boinc_checkpoint_completed(); // Checkpointing completed
        }
        //Update boinc client with percentage
        double frac = (double)(s+1 - block_min) / (double)(block_max - block_min);
        boinc_fraction_done(frac);

        #endif
        for (unsigned long long i = 0; i < blocks * threads; i++){
            if(out[i] > 0){
			    fprintf(seedsout,"%llu\n", out[i]);
                out[i] = 0;
                //Grab values from `out` buffer and print to seedsout
                //Set to 0 after to reset
            }

		}
		fflush(seedsout);

    }


    /*
        The end. This prints speed information to stderr.txt - which will be uploaded to the BOINC server, or it can be reviewed locally in a standsalone run.
    */
    auto end = high_resolution_clock::now();
    auto duration = duration_cast<milliseconds>(end - start);
    checked = blocks*threads*(block_max - block_min);
    fprintf(stderr, "checked = %" PRIu64 "\n", checked);
    fprintf(stderr, "time taken = %f\n", (double)duration.count()/1000.0);

	double seeds_per_second = checked / ((double)duration.count()/1000.0);
	double speedup = seeds_per_second / 199000;
	fprintf(stderr, "seeds per second: %f\n", seeds_per_second);
	fprintf(stderr, "speedup: %fx\n", speedup);
    boinc_finish(0);
}
